#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/ssim_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void CudaGaussConvolveHelper(const int nthreads,
    const Dtype* const in_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, 
    const double* const gauss_kernel, Dtype* const out_data ) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int wstart = pw * stride_w;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    Dtype aveval = 0;
    const Dtype* const in_slice =
        in_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += gauss_kernel[(h-hstart)*kernel_w_+(w-wstart)] * in_slice[h * width + w];
      }
    }
    out_data[index] = aveval;
  }
}

template <typename Dtype>
void SSIMLossLayer<Dtype>::CudaGaussConvolveHelper(const Blob<Dtype>>& in,
    Blob<Dtype>& out) {	
  //Parallelized on the # of outputs to be produced
  CudaGaussConvolveHelper<Dtype><<<CAFFE_GET_BLOCKS(out.count(), CAFFE_CUDA_NUM_THREADS>>>(
        out.count(), in.gpu_data(), in.num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, gauss_kernel_.gpu_data(),
	out.mutable_gpu_data() 
	);
}
template <typename Dtype>
void SSIMLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {	
  int count = bottom[0]->count();
  CudaGaussConvolveHelper(*bottom[0],ux_);
  CudaGaussConvolveHelper(*bottom[0],uy_);

  Blob<Dtype> tempContainer1, tempContainer2;
  tempContainer1.ReshapeLike(*bottom[0]);
  caffe_gpu_sqr(count, bottom[0]->gpu_data(), tempContainer1.mutable_gpu_data()); 
  CudaGaussConvolveHelper(tempContainer1,sx2_);
  caffe_gpu_sqr(count, bottom[1]->gpu_data(), tempContainer1.mutable_gpu_data()); 
  CudaGaussConvolveHelper(tempContainer1,sy2_);
  caffe_gpu_mul(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(), tempContainer1.mutable_gpu_data()); 
  CudaGaussConvolveHelper(tempContainer1,sxy_);

  tempContainer1.ReshapeLike(ux_);
  tempContainer2.ReshapeLike(uy_);
  count = tempContainer1.count();

  //Compute ux^2 and uy^2 and collect ux^2+uy^2 for later use
  caffe_gpu_sqr(count, ux_.gpu_data(), tempContainer1.mutable_gpu_data());
  caffe_gpu_sub(count, sx2_.gpu_data(), tempContainer1.gpu_data(), sx2_.mutable_gpu_data());
  caffe_gpu_sqr(count, uy_.gpu_data(), tempContainer2.mutable_gpu_data());
  caffe_gpu_sub(count, sy2_.gpu_data(), tempContainer2.gpu_data(), sy2_.mutable_gpu_data());
  caffe_gpu_add(count, tempContainer1.gpu_data(), tempContainer2.gpu_data(), tempContainer2.mutable_gpu_data());

  caffe_gpu_mul(count, ux_.gpu_data(), uy_.gpu_data(), tempContainer1.mutable_gpu_data());
  caffe_gpu_sub(count, sxy_.gpu_data(), tempContainer1.gpu_data(), sxy_.mutable_gpu_data());
  
  const Dtype C1 = c1_;
  caffe_gpu_scale(count, Dtype(2), tempContainer1.gpu_data(), tempContainer1.mutable_gpu_data());
  caffe_gpu_add_scalar(count, C1, tempContainer1.mutable_gpu_data());
  caffe_gpu_add_scalar(count, C1, tempContainer2.mutable_gpu_data());
  caffe_gpu_div(count, tempContainer1.gpu_data(), tempContainer2.gpu_data(), lp_.mutable_gpu_data());

  const Dtype C2 = c2_;
  caffe_gpu_add(count, sx2_.gpu_data(), sy2_.gpu_data(), tempContainer2.mutable_gpu_data()); 
  caffe_gpu_add_scalar(count, C2, tempContainer2.mutable_gpu_data());
  caffe_gpu_axpby(count, Dtype(2), sxy_.gpu_data(), Dtype(0), tempContainer1.mutable_gpu_data());
  caffe_gpu_add_scalar(count, C2, tempContainer1.mutable_gpu_data());
  caffe_gpu_div(count, tempContainer1.gpu_data(), tempContainer2.gpu_data(), cs_.mutable_gpu_data());
  
  Dtype ssim = caffe_gpu_dot(count, lp_.gpu_data(),cs_.gpu_data()) / bottom[0]->num();
  Dtype loss = Dtype(1)-ssim;
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void SSIMBackward(const int nthreads, 
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const Dtype* const x,const Dtype* const y,
    const Dtype* const ux,const Dtype* const uy,
    const Dtype* const sx2, const Dtype* const sy2, const Dtype* const sxy,
    const Dtype* const lp, const Dtype* const cs, 
    const double* const gauss_kernel,  Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width ;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    const int q = h* width + w; 
    Dtype gradient = 0;
    const Dtype* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
	int p = ph * pooled_width_ + pw;
        int hstart = ph * stride_h ;
        int wstart = pw * stride_w ;
        int hend = min(hstart + kernel_h, height );
        int wend = min(wstart + kernel_w, width);
	Dtype deriv1 = Dtype(2) * cs[p] * (uy[p]-ux[p]*lp[p]) / (ux[p]*ux[p]+uy[p]*uy[p]+Dtype(c1_));
	Dtype deriv2 = Dtype(2) * lp[p]  / (sx2[p]+sy2[p]+Dtype(c2_));
        gradient += top_diff_slice[p] ;
      }
    }
    bottom_diff[index] = gradient;
  }
}
template <typename Dtype>
void SSIMLossLayer<Dtype>::Backward_cpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  caffe_set(bottom[0]->count(), Dtype(0), bottom_diff);
  const Dtype alpha = -top[0]->cpu_diff()[0] / bottom[0]->num();

  // The main loop
  for (int n = 0; n < bottom[0]->num(); ++n) {
    for (int c = 0; c < channels_; ++c) {
      for (int ph = 0; ph < pooled_height_; ++ph) {
         for (int pw = 0; pw < pooled_width_; ++pw) {
	    int hstart = ph * stride_h_ ;
	    int wstart = pw * stride_w_ ;
	    int hend = min(hstart + kernel_h_, height_);
	    int wend = min(wstart + kernel_w_, width_ );
	    int p = ph * pooled_width_ + pw;
	    Dtype deriv1 = Dtype(2) * cs[p] * (uy[p]-ux[p]*lp[p]) / (ux[p]*ux[p]+uy[p]*uy[p]+Dtype(c1_));
	    Dtype deriv2 = Dtype(2) * lp[p]  / (sx2[p]+sy2[p]+Dtype(c2_));
	    for (int h = hstart; h < hend; ++h) {
	      for (int w = wstart; w < wend; ++w) {
		int q = h * width_ + w;
		bottom_diff[q] += 
                    gaussian[(h-hstart)*kernel_w_+(w-wstart)]* ( deriv1 + ( deriv2 * ((y[q] - uy[p]) - cs[p]*(x[q]-ux[p]))));
	      }
	    }
	  }
	}
	// offset
	x+= bottom[0]->offset(0,1);
	y+= bottom[1]->offset(0,1);
	bottom_diff += bottom[0]->offset(0, 1);
	ux+= ux_.offset(0,1);
	uy+= ux_.offset(0,1);
	sx2+= ux_.offset(0,1);
	sy2+= ux_.offset(0,1);
	sxy+= ux_.offset(0,1);
	lp+= ux_.offset(0,1);
	cs+= ux_.offset(0,1);
    }
  }
  caffe_cpu_scale(
          bottom[0]->count(),              // count
          alpha,                              // alpha
          bottom[0]->cpu_diff(),              // x
          bottom_diff);  // y
}

template <typename Dtype>
void SSIMLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_sign(bottom[i]->count(), diff_.gpu_data(),
      		     bottom[i]->mutable_gpu_diff());
      caffe_gpu_scale(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          bottom[i]->gpu_diff(),                   // x
          bottom[i]->mutable_gpu_diff());  // y
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SSIMLossLayer);

}  // namespace caffe
